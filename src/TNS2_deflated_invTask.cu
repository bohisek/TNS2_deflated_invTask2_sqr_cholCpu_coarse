#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cstring>
#include <iomanip>
#include <numeric>
#include <stdlib.h>
#include <math.h>

#define min(a,b) a<b ? a : b
#define max(a,b) a>b ? a : b
#define DEFL

using namespace std;

typedef double T;

typedef struct {
	unsigned int blocks;
	unsigned int blockSize;
	unsigned int blocksZ;
	unsigned int Nx;
	unsigned int Ny;
	unsigned int size;
	unsigned int steps;
	unsigned int maxiter;
	T dt;
	T maxres;
	unsigned int nRowsZ;
	unsigned int nDV;
    unsigned int NxZ;
} Parameters;


typedef struct {
	T rho;
	T cp;
	T k;
} MaterialProperties;

Parameters params;

MaterialProperties steel;
MaterialProperties Ag;
MaterialProperties MgO;
MaterialProperties inconel;
MaterialProperties NiCr;

#include "cpuFunctions.h"
#include "cpuFunctionsDeflation.h"
#include "cudaFunctions.h"
#include "cudaFunctionsDeflation.h"

int main(void)
{
    // Parameters
	params.blocks = 16;  
	params.blockSize = 128;  
	params.Nx  = 32;
	params.Ny  = 128;
	params.size = params.Nx*params.Ny + 2*params.Nx;
	params.steps  = 300;
	params.maxiter  = 1000000;
	params.dt = 1./300.;    // 0.0033333333333333;
	params.maxres  = 1e-4;
	params.nRowsZ = 4;    // number of rows/columns for one deflation vector
	params.nDV = (params.Nx*params.Ny) / (params.nRowsZ*params.nRowsZ);
	params.NxZ = params.Nx/params.nRowsZ; // number of course cells in a row
		
	// steel
	steel.rho = 7610.0; // 7700
    steel.cp  = 545.0; // 560
    steel.k   = 21.0;

    // Ag
    Ag.rho = 8957.0;
    Ag.cp  = 362.0; // 368
    Ag.k   = 120; // 111.5

    // MgO
    MgO.rho = 3150.0;
    MgO.cp  = 1110.0; // 1140
    MgO.k   = 11.5;   // 10
    
    // inconel
    inconel.rho = 8470.0;
    inconel.cp  = 500.0; // 520
    inconel.k   = 20.5; 
    
    // NiCr
    NiCr.rho = 8200.0;
    NiCr.cp  = 528.0;
    NiCr.k   = 24.5;
    
    T t = 0; // time
    T totalIter = 0;
    
    
    dim3 dimGrid(params.blocks);
    dim3 dimBlock(params.blockSize);
    	
	cout << "example 1: inverse task (Pohanka)" << endl;
	cout << "nDV: " << params.nDV << ", params.NxZ: " << params.NxZ << endl;
	
	cpuInit(params.blocks, params.Nx, params.Ny);
	cpuInitDeflation(params.Nx, params.Ny, params.NxZ, params.nDV);
	
	readGeometry(hm, params.Nx, params.Ny);	// materials
	
	readCoords(xc, dx, "xCoords.txt");
	readCoords(yc, dy, "yCoords.txt");
	
	readBC(tHF, params.steps);
	
	initX(hT, params.Nx, params.Ny);
	initA(params.dt, params.Nx, params.Ny);
	
	initAZ(params.Nx, params.Ny, params.nRowsZ);
	initE(params.Nx, params.Ny, params.nRowsZ, params.NxZ);
	spChol(params.NxZ, params.nDV);
	
	//check_varZ(ecc, ess, params.nDV, params.NxZ);
	
	//saveData<int>(hm, "materials1024", params.Nx, params.Ny);
	//saveDataInTime(hT, t, "temperature1024_res1e-3_TNS1_cpuICHOL", params.Nx, params.Ny);
	
	//check_var(dy, params.Nx, params.Ny);

	// CUDA
	cudaInit(hT, hV, hcc, hss, hww, hqB, params.blocks, params.Nx, params.Ny);
	cudaInitDeflation(azc, azw, azs, ecc, eww, ess, params.NxZ, params.nDV, params.Nx, params.Ny);
	
	makeTNS2<<<32,128>>>(kc, ks, kw, ksw, kse, kss, kww, ksee, ksse, dcc, dss, dww, params.Nx, params.Ny);

	
	hipEvent_t startT, stopT;
	float elapsedTime;
	hipEventCreate(&startT);
	hipEventCreate(&stopT);
	hipEventRecord(startT,0);

	for (int miter=0; miter<params.steps; miter++) {

		hipMemcpy(dr, dT, sizeof(T)*params.size, hipMemcpyDeviceToDevice);	// r = rhs
		elementWiseMul<<<32,128>>>(dr, dV, params.Nx);	          // r = V*r
		
		// add Neumann boundary here ... r = r + NeumannBC (dqB)
		//addNeumannBC<<<1,32>>>(dr, dqB, (T)-1.0e6, params.Nx);    // constant
		addNeumannBC<<<1,32>>>(dr, dqB, tHF[miter], params.Nx);     // time dependent

		SpMVv1<<<32,128>>>(dq, dcc, dss, dww, dT, params.Nx);      // q = Ax (version 1)
		AXPY<<<32,128>>>(dr, dq, (T)-1., (T)1., params.Nx);        // r = r - q
		
#ifdef DEFL
		
		// ---  (DEFLATION) r = Pr  ---
		///hipMemset(dyZ,0,sizeof(T)*(params.nDV+2*params.NxZ)); // reset;
		ZTransXYDeflation<<<4,64>>>(drZ, dr, params.nRowsZ, params.NxZ, params.Nx);  // y1 = Z'*y
		// E*y2 = y1 (begin)
		hipMemcpy(hrZ, drZ, (params.nDV+2*params.NxZ)*sizeof(double), hipMemcpyDeviceToHost);   // copy drZ to hrZ
		solve(params.NxZ,params.nDV);
		hipMemcpy(dyZ, hyZ, (params.nDV+2*params.NxZ)*sizeof(double), hipMemcpyHostToDevice);   //copy hyZ to dyZ
		///printX(hyZ, params.NxZ,params.nDV);
		// E*y2 = y1 (end)
		YMinusAzXYDeflation<<<32,128>>>(dr,dyZ,dazc,dazw,dazs,params.nRowsZ,params.NxZ,params.Nx);  // r = P*r
#endif	    

		SpMVv2<<<32,128>>>(dz, kc, ks, kw, ksw, kse, kss, kww, ksee, ksse, dr, params.Nx, params.Ny);   // z = M^(-1)r (version 2)
		DOTGPU<T,128><<<dimGrid,dimBlock,params.blockSize*sizeof(T)>>>(drh, dr, dz, params.Nx, params.Ny);
		hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
		rhNew = dot(hrh,params.blocks);
		
		
		// --- stop criterion here ---
		DOTGPU<T,128><<<dimGrid,dimBlock,params.blockSize*sizeof(T)>>>(drh, dz, dz, params.Nx, params.Ny);
		hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
		stop = dot(hrh,params.blocks) * params.maxres * params.maxres;
		
		iter = 0;
		
		//cout << "stop:" << stop << ", residual: " << rhNew << endl;
		
		while (rhNew > stop && iter < params.maxiter) {
		//while (iter < 40) {  // only testing

			iter++;
			totalIter++;
			//cout << "iteration:" << iter << ", residual: " << rhNew << endl;
			//cout << rhNew << endl;

			if (iter==1) {
				hipMemcpy(dp, dz, sizeof(T)*params.size,hipMemcpyDeviceToDevice);
			}
			else {
				bt = rhNew/rhOld;	
				AXPY<<<32,128>>>(dp, dz, (T)1., bt, params.Nx);   // p = z + beta*p	
			}

			SpMVv1<<<32,128>>>(dq, dcc, dss, dww, dp, params.Nx);  // q = Ap (version 1)
			
#ifdef DEFL
			
			// ---  (DEFLATION) q = Pq  ---
			ZTransXYDeflation<<<4,64>>>(drZ, dq, params.nRowsZ, params.NxZ, params.Nx);  // y1 = Z'*y
			// E*y2 = y1 (begin)
			hipMemcpy(hrZ, drZ, (params.nDV+2*params.NxZ)*sizeof(T), hipMemcpyDeviceToHost);   // copy drZ to hrZ
			solve(params.NxZ,params.nDV);
			hipMemcpy(dyZ, hyZ, (params.nDV+2*params.NxZ)*sizeof(T), hipMemcpyHostToDevice);   //copy hyZ to dyZ
			// E*y2 = y1 (end)
			YMinusAzXYDeflation<<<32,128>>>(dq,dyZ,dazc,dazw,dazs,params.nRowsZ,params.NxZ,params.Nx);  // q = Pq
#endif



			DOTGPU<T,128><<<dimGrid,dimBlock,params.blockSize*sizeof(T)>>>(dsg, dp, dq, params.Nx, params.Ny);   // sigma = <p,q>
			//hipMemcpy(&sg, dsg, 1*sizeof(T), hipMemcpyDeviceToHost);
			//hipMemset(dsg,0,sizeof(T)); // reset;
			hipMemcpy(hsg, dsg, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
			sg = dot(hsg,params.blocks);
			ap = rhNew/sg;	// alpha = rhoNew / sigma
			AXPY<<<32,128>>>(dr, dq, -ap, (T)1., params.Nx);   // r = r - alpha*q
			AXPY<<<32,128>>>(dT, dp,  ap, (T)1., params.Nx);   // x = x + alpha*p

			SpMVv2<<<32,128>>>(dz, kc, ks, kw, ksw, kse, kss, kww, ksee, ksse, dr, params.Nx, params.Ny);   // z = M^(-1)r (version 2)

			rhOld = rhNew;

			DOTGPU<T,128><<<dimGrid,dimBlock,params.blockSize*sizeof(T)>>>(drh, dr, dz, params.Nx, params.Ny);   // rhoNew = <r,z>		
			//hipMemcpy(&rhNew, drh, 1*sizeof(T), hipMemcpyDeviceToHost);
			//hipMemset(drh,0,sizeof(T));   // reset;
			hipMemcpy(hrh, drh, params.blocks*sizeof(T), hipMemcpyDeviceToHost);
		    rhNew = dot(hrh,params.blocks);
		}
		
		// x = x +  (deflation)
		
		t += params.dt;
		//cout << endl << "time: " << t << " ,timestep:" << miter << " ,iteration:" << iter << endl;
		
		//if ((miter+1)%4000==0)
		//{
		//hipMemcpy(hT, dT, sizeof(T)*params.size, hipMemcpyDeviceToHost);
		//saveDataInTime(hT, t, "temperature_res1e-4_TNS1float", params.Nx, params.Ny);
		//}

	}
	
	hipEventRecord(stopT,0);
	hipEventSynchronize(stopT);
	hipEventElapsedTime(&elapsedTime, startT, stopT);
	cout<< "ellapsed time (cuda): " << elapsedTime << " miliseconds"	<< endl;
	
	cout << "Simulation finished." << endl;
	cout << "total number of iterations: " << totalIter << endl;
	
	hipEventDestroy(startT);
	hipEventDestroy(stopT);
	
	cudaFinalizeDeflation();
	cudaFinalize();
	
	cpuFinalizeDeflation();
	cpuFinalize();
	
	return 0;
}

